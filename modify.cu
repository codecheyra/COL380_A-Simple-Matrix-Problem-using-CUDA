#include "hip/hip_runtime.h"
#include "modify.cuh"
#include <hip/hip_runtime.h>
#include <vector>
#include <cassert>

using std::vector;

__global__ void countFreqKernel(const int *d_input, int total_elements, int *d_freq, int range)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_elements)
    {
        int val = d_input[idx];
        atomicAdd(&d_freq[val], 1);
    }
}

__device__ int binarySearch(const int *d_prefix, int range, int idx)
{
    int low = 1;
    int high = range + 1;
    while (low < high)
    {
        int mid = (low + high) >> 1;
        if (d_prefix[mid] <= idx)
            low = mid + 1;
        else
            high = mid;
    }
    return low - 1;
}

__global__ void assignSortedKernel(const int *d_prefix, int range, int total_elements, int *d_output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_elements)
    {
        int value = binarySearch(d_prefix, range, idx);
        d_output[idx] = value;
    }
}

vector<vector<vector<int>>> modify(vector<vector<vector<int>>> &matrices, vector<int> &ranges)
{
    int numMatrices = matrices.size();
    vector<vector<vector<int>>> result(numMatrices);
    vector<hipStream_t> streams(numMatrices);
    for (int k = 0; k < numMatrices; k++)
    {
        hipStreamCreate(&streams[k]);
    }
    for (int k = 0; k < numMatrices; k++)
    {
        int rows = matrices[k].size();
        assert(rows > 0);
        int cols = matrices[k][0].size();
        int total_elements = rows * cols;
        int range = ranges[k];
        vector<int> h_input(total_elements);
        for (int i = 0; i < rows; i++)
        {
            for (int j = 0; j < cols; j++)
            {
                h_input[i * cols + j] = matrices[k][i][j];
            }
        }
        int *d_input;
        int *d_output;
        int *d_freq;
        int *d_prefix;
        hipMalloc(&d_input, total_elements * sizeof(int));
        hipMalloc(&d_output, total_elements * sizeof(int));
        hipMalloc(&d_freq, (range + 2) * sizeof(int));
        hipMalloc(&d_prefix, (range + 2) * sizeof(int));
        hipMemsetAsync(d_freq, 0, (range + 2) * sizeof(int), streams[k]);
        hipMemcpyAsync(d_input, h_input.data(), total_elements * sizeof(int), hipMemcpyHostToDevice, streams[k]);
        int threadsPerBlock = 256;
        int blocks = (total_elements + threadsPerBlock - 1) / threadsPerBlock;
        countFreqKernel<<<blocks, threadsPerBlock, 0, streams[k]>>>(d_input, total_elements, d_freq, range);
        vector<int> h_freq(range + 2, 0);
        hipMemcpyAsync(h_freq.data(), d_freq, (range + 2) * sizeof(int), hipMemcpyDeviceToHost, streams[k]);
        hipStreamSynchronize(streams[k]);
        vector<int> h_prefix(range + 2, 0);
        h_prefix[1] = 0;
        for (int v = 1; v <= range; v++)
        {
            h_prefix[v + 1] = h_prefix[v] + h_freq[v];
        }
        hipMemcpyAsync(d_prefix, h_prefix.data(), (range + 2) * sizeof(int), hipMemcpyHostToDevice, streams[k]);
        blocks = (total_elements + threadsPerBlock - 1) / threadsPerBlock;
        assignSortedKernel<<<blocks, threadsPerBlock, 0, streams[k]>>>(d_prefix, range, total_elements, d_output);
        vector<int> h_output(total_elements);
        hipMemcpyAsync(h_output.data(), d_output, total_elements * sizeof(int), hipMemcpyDeviceToHost, streams[k]);
        hipStreamSynchronize(streams[k]);
        vector<vector<int>> sorted_matrix(rows, vector<int>(cols));
        for (int i = 0; i < rows; i++)
        {
            for (int j = 0; j < cols; j++)
            {
                sorted_matrix[i][j] = h_output[i * cols + j];
            }
        }
        result[k] = sorted_matrix;
        hipFree(d_input);
        hipFree(d_output);
        hipFree(d_freq);
        hipFree(d_prefix);
    }
    for (int k = 0; k < numMatrices; k++)
    {
        hipStreamDestroy(streams[k]);
    }
    return result;
}

